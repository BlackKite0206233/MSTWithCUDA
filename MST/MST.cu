#include "hip/hip_runtime.h"
#pragma once
#include <cmath>
#include "MST.h"




MST::MST(int _vNum, int _eNum) : count(0) {
    this->vNum = abs(_vNum);
    this->eNum = abs(_eNum);

    this->v = new int[this->vNum];
    this->p = new int[this->vNum];

    this->e = new Edge[this->eNum];
}

MST::~MST() {
    delete[] this->v;
    delete[] this->p;
    delete[] this->e;
}

void MST::addEdge(int _x, int _y, int _weight) {
    if(count < this->eNum && _x >= 0 && _x < this->vNum && _y >= 0 && _y < this->vNum) {
        this->e[count].x = _x;
        this->e[count].y = _y;
        this->e[count].weight = _weight;
        count++;
    }
}

void MST::cal() {

}

ofstream& operator <<(ofstream& _ofs, MST& _smt) {
    for(int i = 0; i < _smt.vNum; i++) 
        _ofs << "point " << i << " connect to point " << _smt.p[i] << endl;

    return _ofs;
}

