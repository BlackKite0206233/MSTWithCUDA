#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cmath>
#include "MST.h"

#define MAX_NUM 100000
#define THREAD_PER_BLOCK 1024

__device__ int find(int x, int *p) {
    return x == p[x] ? x : (p[x] = find(p[x], p));
}

__device__ void Union(int x, int y, int *p) {
    p[find(x, p)] = find(y, p);
}

__global__ void init(int *d, int *e) {
    long long int index = threadIdx.x + blockIdx.x * blockDim.x;
    d[index] = MAX_NUM;
    e[index] = 0;
}

__global__ void smt(int *cross_dege, int *d, int *e, int *p, Edge *edge) {
    long long int index = threadIdx.x + blockIdx.x * blockDim.x;
    int x = find(edge[index].x, p);
    int y = find(edge[index].y, p);
    int weight = edge[index].weight;
    if(x != y) {
        (*cross_dege)++;
        if(weight < d[x] || weight == d[x] && index < e[x])
            d[x] = weight, e[x] = index;
        if(weight < d[y] || weight == d[y] && index < e[y])
            d[y] = weight, e[y] = index;
    }
   
}

__global__ void merge(int *d, int *e, int *p, Edge *edge) {
    long long int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(d[index] != MAX_NUM)
        Union(edge[e[index]].x, edge[e[index]].y, p);
}


MST::MST(int _vNum, int _eNum) : count(0) {
    this->vNum = abs(_vNum);
    this->eNum = abs(_eNum);

    this->p = new int[this->vNum];
    for(int i = 0; i < this->vNum; i++)
        this->p[i] = i;

    this->edge = new Edge[this->eNum];
}

MST::~MST() {
    delete[] this->p;
    delete[] this->edge;
}

void MST::addEdge(int _x, int _y, int _weight) {
    if(count < this->eNum && _x >= 0 && _x < this->vNum && _y >= 0 && _y < this->vNum) {
        this->edge[count].x = _x;
        this->edge[count].y = _y;
        this->edge[count].weight = _weight;
        count++;
    }
}

void MST::cal() {
    int *v_d, *e_d, *p_d;
    Edge *edge_d;

    int *v = new int[this->vNum];
    int *e = new int[this->vNum];

    hipMalloc((int **)&v_d, sizeof(int) * this->vNum);
    hipMalloc((int **)&e_d, sizeof(int) * this->vNum);
    hipMalloc((int **)&p_d, sizeof(int) * this->vNum);
    hipMalloc((Edge **)&edge_d, sizeof(Edge) * this->eNum);

    hipMemcpy(p_d, this->p, sizeof(int) * this->vNum, hipMemcpyHostToDevice);
    hipMemcpy(edge_d, this->edge, sizeof(Edge) * this->eNum, hipMemcpyHostToDevice);

    while(true) {
        int cross_edge = 0;

        init<<<this->vNum / THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK>>>(v_d, e_d);

        
        hipMemcpy(v, v_d, sizeof(int) * this->vNum, hipMemcpyDeviceToHost);
        hipMemcpy(e, e_d, sizeof(int) * this->vNum, hipMemcpyDeviceToHost);
        hipMemcpy(this->p, p_d, sizeof(int) * this->vNum, hipMemcpyDeviceToHost);

        for(int i = 0; i < this->vNum; i++) 
            cout << v[i] << " " << e[i] << " " << this->p[i] << endl;
            

        smt<<<this->vNum / THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK>>>(&cross_edge, v_d, e_d, p_d, edge_d);

        if(cross_edge == 0)
            break;

        merge<<<this->vNum / THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK>>>(v_d, e_d, p_d, edge_d);
    }

    hipMemcpy(this->p, p_d, sizeof(int) * this->vNum, hipMemcpyDeviceToHost);
    
    hipFree(v_d);
    hipFree(e_d);
    hipFree(p_d);
    hipFree(edge_d);
}

ofstream& operator <<(ofstream& _ofs, MST& _smt) {
    for(int i = 0; i < _smt.vNum; i++) 
        _ofs << "point " << i << " connect to point " << _smt.p[i] << endl;

    return _ofs;
}

